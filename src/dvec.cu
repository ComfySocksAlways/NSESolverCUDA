#include "dvec.h"

DVec::DVec(int N){
    checkErr(hipMalloc((&data), sizeof(double) * N));
    checkErr(hipMemset(data,0,sizeof(double) * N));
    hipsparseCreateDnVec(&vecdescr, N, data, HIP_R_64F);
    ndata = N;
    
    // Construction from vector doesn't have matrix meta data
    nr = -1;
    nc = -1; 
}

DVec::DVec(const std::vector<double>& input){
    checkErr(hipMalloc((&data), sizeof(double) * input.size()));
    checkErr(hipMemcpy(data, input.data(), sizeof(double) * input.size(), hipMemcpyHostToDevice));
    hipsparseCreateDnVec(&vecdescr, input.size(), data, HIP_R_64F);
    ndata = input.size();
    
    // Construction from vector doesn't have matrix meta data
    nr = -1;
    nc = -1;
}

DVec::DVec(const DMatrix& input){
    hipsparseCreateDnVec(&vecdescr,input.nc*input.nr,input.Md+2,HIP_R_64F);
}

DVec::DVec(const DVec &other){
    checkErr(hipMalloc((&data), sizeof(double) * other.ndata));
    checkErr(hipMemcpy(data, other.data, sizeof(double) * other.ndata, hipMemcpyDeviceToDevice));
    hipsparseCreateDnVec(&vecdescr, other.ndata, data, HIP_R_64F);
    ndata = other.ndata;
    nr = other.nr;
    nc = other.nc;
}


void DVec::DVecToHost(std::vector<double>& output){
    size_t size = sizeof(double) * ndata;
    output.reserve(size);
    checkErr(hipMemcpy(output.data(), data, size, hipMemcpyDeviceToHost));
}


