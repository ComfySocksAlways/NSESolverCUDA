#include "dmatrix.h"

DMatrix::DMatrix(const Matrix<double> & M){
    size_t size = sizeof(double) * (M.nc * M.nr + 2);
    checkErr(hipMalloc(&Md,size));
    double dnr = static_cast<double>(M.nr);
    double dnc = static_cast<double>(M.nc);
    hipMemcpy(Md, &dnr, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Md + 1, &dnc, sizeof(double), hipMemcpyHostToDevice);
    checkErr(hipMemcpy(Md + 2, M.data,  M.nr * M.nc * sizeof(double) , hipMemcpyHostToDevice));
    // Cpy Meta data
    nr = M.nr;
    nc = M.nc;
}


void DMatrix::deviceMatrixToHost(Matrix<double> &M){
    // hipMemcpy(&nr, Md, sizeof(double), hipMemcpyDeviceToHost);
    // hipMemcpy(&nc, Md + 1, sizeof(double), hipMemcpyDeviceToHost);
    if((static_cast<int>(nr) == M.nr ) && static_cast<int>(nc) == M.nc)
        hipMemcpy(M.data, Md + 2, M.nr * M.nc * sizeof(double), hipMemcpyDeviceToHost);
    else
        std::cout << "Size mismatch, could not copy" << std::endl;
}