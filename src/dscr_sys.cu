#include "dscr_sys.h"


hipsparseHandle_t DCSRSystem::handle;
bool DCSRSystem::handleInitialized = false;
// DCSRSystem::DCSRSystem(){

// }

DCSRSystem::DCSRSystem(const Matrix<double> &A){
    CSR csrM(A);
    // Coeff Matrix Alloc
    checkErr(hipMalloc(&dvals, sizeof(double) * csrM.nnz));
    checkErr(hipMalloc(&dcolptr, sizeof(int) * csrM.nnz));
    checkErr(hipMalloc(&drowptr, sizeof(int) * (csrM.nr+1)));

    // Copy Data
    checkErr(hipMemcpy(dvals, csrM.vals.data(), sizeof(double) * csrM.nnz, hipMemcpyHostToDevice));
    checkErr(hipMemcpy(dcolptr, csrM.colptr.data(), sizeof(int) * csrM.nnz, hipMemcpyHostToDevice));
    checkErr(hipMemcpy(drowptr, csrM.rowptr.data(), sizeof(int) * (csrM.nr+1), hipMemcpyHostToDevice));

    // cuSparse Handles
    if(!handleInitialized){
        hipsparseCreate(&handle);
        handleInitialized = true;
        std::atexit([]() {
            hipsparseDestroy(handle);
        });
    }
    hipsparseCreateConstCsr(     &descr,
                                csrM.nr,
                                csrM.nc,
                                csrM.nnz,
                                drowptr,
                                dcolptr,
                                dvals,
                                HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIP_R_64F
                        );
}   

void DCSRSystem::DCSRSpMV(DVec &x, DVec& y, double alpha, double beta){
 size_t bufferSize;
 CHECK_CUSPARSE(   hipsparseSpMV_bufferSize(handle,
                            hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha,
                            descr,
                            x.vecdescr,
                            &beta,
                            y.vecdescr,
                            HIP_R_64F,
                            HIPSPARSE_SPMV_CSR_ALG1,
                            &bufferSize
                            ));
    void *externalBuffer;
    hipMalloc(&externalBuffer, bufferSize);
    // ToDo 1: Ensure this preprocess is only called once, for multi calls to this product.
    // ToDo 2: Maybe make x of const type?
    // hipsparseSpMV_preprocess(handle,
    //                         hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                         &alpha,
    //                         descr,
    //                         x.vecdescr,
    //                         &beta,
    //                         y.vecdescr,
    //                         HIP_R_64F,
    //                         HIPSPARSE_SPMV_CSR_ALG1,
    //                         externalBuffer);
CHECK_CUSPARSE(    hipsparseSpMV(   handle,
                    hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    descr,
                    x.vecdescr,
                    &beta,
                    y.vecdescr,
                    HIP_R_64F,
                    HIPSPARSE_SPMV_CSR_ALG1,
                    externalBuffer));
    hipFree(externalBuffer);
}


void DCSRSystem::MakeBuffer(DVec &x, DVec &y, double alpha, double beta){
    size_t bufferSize;
    CHECK_CUSPARSE(   hipsparseSpMV_bufferSize(handle,
                            hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha,
                            descr,
                            x.vecdescr,
                            &beta,
                            y.vecdescr,
                            HIP_R_64F,
                            HIPSPARSE_SPMV_CSR_ALG1,
                            &bufferSize
                            ));
    hipMalloc(&buffer, bufferSize);
}

void DCSRSystem::PreProcess(DVec &x, DVec &y, double alpha, double beta){
    CHECK_CUSPARSE( hipsparseSpMV_preprocess( handle,
                    hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    descr,
                    x.vecdescr,
                    &beta,
                    y.vecdescr,
                    HIP_R_64F,
                    HIPSPARSE_SPMV_CSR_ALG1,
                    buffer));   
}

void DCSRSystem::Axplusy(DVec &x, DVec &y, double alpha, double beta){
    CHECK_CUSPARSE(    hipsparseSpMV(   handle,
                    hipsparseOperation_t::HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    descr,
                    x.vecdescr,
                    &beta,
                    y.vecdescr,
                    HIP_R_64F,
                    HIPSPARSE_SPMV_CSR_ALG1,
                    buffer));
}


void DCSRSystem::ClearBuffer(){
    hipFree(buffer);
}